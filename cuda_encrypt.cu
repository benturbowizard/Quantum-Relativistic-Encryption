#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void encrypt_kernel(const uchar* plaintext, uchar* ciphertext, 
                                uint8_t* key, int length) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < length) {
    ciphertext[i] = plaintext[i] ^ key[i]; 
  }

}

void encrypt(uchar* plaintext, uchar* ciphertext, uint8_t* key, int length) {

  uchar* device_plaintext;
  uchar* device_ciphertext;
  uint8_t* device_key;
  
  // Allocate GPU memory
  hipMalloc(&device_plaintext, length);
  hipMalloc(&device_ciphertext, length);
  hipMalloc(&device_key, length);

  // Copy data to GPU
  hipMemcpy(device_plaintext, plaintext, length, 
             hipMemcpyHostToDevice);
  hipMemcpy(device_key, key, length, 
             hipMemcpyHostToDevice);
             
  // Launch encryption kernel
  dim3 threads(256);
  dim3 blocks((length + threads.x - 1) / threads.x);
  encrypt_kernel<<<blocks, threads>>>(device_plaintext, device_ciphertext, 
                                      device_key, length);

  // Copy encrypted data back to CPU
  hipMemcpy(ciphertext, device_ciphertext, length, 
             hipMemcpyDeviceToHost);

  // Free GPU memory
  hipFree(device_plaintext);
  hipFree(device_ciphertext);
  hipFree(device_key);

}